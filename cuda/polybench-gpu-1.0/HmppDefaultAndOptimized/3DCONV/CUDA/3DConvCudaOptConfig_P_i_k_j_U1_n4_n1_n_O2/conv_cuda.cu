// ** Original codelet code **
//
// #pragma hmppcg cpiparam __arg0 IN A%hmpp_codelet__conv3D: (1, 0)
// #pragma hmppcg cpiparam __arg1 INOUT B%hmpp_codelet__conv3D: (1, 1)
// 
// #pragma hmppcg cpicall hmpp_codelet__conv3D(__arg0, __arg1): 1
// 
// 
// /* begin of extracted source code for directive set "conv" */
// 
// 
// # 29 "threedconv.c"
// typedef float  DATA_TYPE;
// 
// 
// # 34 "threedconv.c"
// void hmpp_codelet__conv3D(DATA_TYPE A[256][256][256], DATA_TYPE B[256][256][256])
// {
//   int  i, j, k;
//   DATA_TYPE  c11, c12, c13, c21, c22, c23, c31, c32, c33;
// 
//   c11 = +2;  c21 = +5;  c31 =  -8;
//   c12 =  -3;  c22 = +6;  c32 =  -9;
//   c13 = +4;  c23 = +7;  c33 = +10;
// 
// #pragma hmppcg grid blocksize 32 X 8
// # 14 "<preprocessor>"
// # 44 "threedconv.c"
// #pragma hmppcg permute i, k, j
// # 17 "<preprocessor>"
// # 46 "threedconv.c"
//   for (i = 1 ; i < 256 - 1 ; ++i)
//     {
// #pragma hmppcg unroll 4, guarded
// # 22 "<preprocessor>"
// # 49 "threedconv.c"
//       for (j = 1 ; j < 256 - 1 ; ++j)
//         {
//           for (k = 1 ; k < 256 - 1 ; ++k)
//             {
// 
// 
// 
// 
// 
// 
// 
// 
// 
//               B[i][j][k] = 0 + c11 * A[i - 1][j - 1][k - 1] + c13 * A[i + 1][j - 1][k - 1] + c21 * A[i - 1][j - 1][k - 1] + c23 * A[i + 1][j - 1][k - 1] + c31 * A[i - 1][j - 1][k - 1] + c33 * A[i + 1][j - 1][k - 1] + c12 * A[i + 0][j - 1][k + 0] + c22 * A[i + 0][j + 0][k + 0] + c32 * A[i + 0][j + 1][k + 0] + c11 * A[i - 1][j - 1][k + 1] + c13 * A[i + 1][j - 1][k + 1] + c21 * A[i - 1][j + 0][k + 1] + c23 * A[i + 1][j + 0][k + 1] + c31 * A[i - 1][j + 1][k + 1] + c33 * A[i + 1][j + 1][k + 1];
//             }
//         }
//     }
// }
// 
// 
// /* end of extracted source code for directive set "conv" */
// 
// 
//
// ** End of original codelet codelet **



#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>


#ifdef _MSC_VER
#  define HMPPCG_RESTRICT
typedef __int8 int8_t;
typedef unsigned __int8 uint8_t;
typedef __int16 int16_t;
typedef unsigned __int16 uint16_t;
typedef __int32 int32_t;
typedef unsigned __int32 uint32_t;
typedef __int64 int64_t;
typedef unsigned __int64 uint64_t;
#  ifdef _WIN64
typedef int64_t intptr_t;
#  else
typedef int32_t intptr_t;
#  endif
#else
#  if defined(__GNUC__) || defined(__RESTRICT)
#    define HMPPCG_RESTRICT __restrict
#  else
#    define HMPPCG_RESTRICT
#  endif
#  include <stdint.h>
#endif

// Dynamic array
typedef struct __hmppcg_array_struct
{
  void *array;
  size_t *size;
  size_t elsize;
} __hmppcg_array_t;

// Data section
typedef struct __hmppcg_DataSection
{
  size_t from;
  size_t to;
  size_t step;
} __hmppcg_DataSection;


#include <hip/hip_runtime.h>

#if CUDART_VERSION < 2000
#error Bad CUDA Runtime version. CUDA Toolkit 2.0+ required.
#endif

#define HMPP_CONSTMEM_OFFSET 0

#include <map>
#include <string>
// ----------------------------------------------------------------------------
// HMPP CUDA support classes
// ----------------------------------------------------------------------------

#ifndef __HMPP_CUDADATA_H__
#define __HMPP_CUDADATA_H__

#ifndef HMPPCG_WARP_SIZE
#define HMPPCG_WARP_SIZE 32
#endif

enum CopyKind
{
  HostToHost  = 0,
  HostToDevice = 1,
  DeviceToHost = 2,
  DeviceToDevice = 3,
};

inline int hmppcg_check_status(const char *file,int line,hipError_t status)
{
  if(status != hipSuccess)
  {
    fprintf(stderr, "%s:%d CUDA Error: %s\n", file, line,
            hipGetErrorString(status));
    return -1;
  }
  return 0;
}


#define CHECK_STATUS(X) hmppcg_check_status(__FILE__,__LINE__,(X))

#define HMPP_CHECK_GRID_BOUNDARY(x) \
   if(x>65535){\
     fprintf(stderr, "%s:%d Grid Dimension Error: '%s' exceeds the 65535 dimension limit. Please modify the grid size configuration (see the hmppcg grid blocksize pragma) or switch to 2D gridification\n", __FILE__,__LINE__, #x);\
     exit(-1) ;\
   }

#define HMPP_CHECK_BLOCK_BOUNDARY(x) \
  if(x > devProp.maxThreadsPerBlock){		\
    fprintf(stderr, "%s:%d Number of threads per block exceeds for the HWA: it is '%d' and HWA supports up to '%d'. Please modify the block size configuration (see the hmppcg grid blocksize pragma)\n", __FILE__,__LINE__, x, devProp.maxThreadsPerBlock); \
    exit(-1) ;								\
  }

// ----------------------------------------------------------------------------
// class DefaultPolicy
// ----------------------------------------------------------------------------

struct DefaultPolicy
{
public:

  DefaultPolicy()
  {
  }

  virtual ~DefaultPolicy()
  {
  }

  int deviceAlloc(void **ptr,size_t size)
  {
    if( CHECK_STATUS(hipStreamCreate(&stream_)) != 0 ) return -1;
    if( CHECK_STATUS(hipMalloc(ptr,size)) != 0 ) return -1;
#if CUDA_VERSION >= 3020
    if( CHECK_STATUS(hipEventCreateWithFlags(&event, hipEventDisableTiming | hipEventBlockingSync)) != 0)
      return -1;
#else
    if( CHECK_STATUS(cudaEventCreateWithFlags(&event, cudaEventBlockingSync)) != 0)
      return -1;
#endif
    return 0;
  }

  int deviceFree(void *ptr)
  {
    if( CHECK_STATUS(hipStreamDestroy(stream_)) != 0) return -1;
    if( CHECK_STATUS(hipFree(ptr)) != 0) return -1;
    if( CHECK_STATUS(hipEventDestroy(event)) != 0) return -1;
    return 0;
  }

  int deviceMemcpy(void *dst,const void *src,size_t size,CopyKind kind,bool async)
  {
    static hipMemcpyKind cudaKind[]
      = {hipMemcpyHostToHost,
         hipMemcpyHostToDevice,
         hipMemcpyDeviceToHost,
         hipMemcpyDeviceToDevice };

    if(async)
    {
      return CHECK_STATUS(hipMemcpyAsync(dst,src,size,cudaKind[kind],stream_));
    }
    else
    {
      return CHECK_STATUS(hipMemcpy(dst,src,size,cudaKind[kind]));
    }
  }

  int makeStreamWait(hipStream_t wstream)
  {
    int status;
    status = CHECK_STATUS(hipEventRecord(event, stream_));
    if (status != 0)
      return status;
#if CUDA_VERSION >= 3020
    return CHECK_STATUS(hipStreamWaitEvent(wstream, event, 0));
#else
    return CHECK_STATUS(cudaEventSynchronize(event));
#endif
  }

  int waitOnEvent(hipEvent_t wevent)
  {
#if CUDA_VERSION >= 3020
    return CHECK_STATUS(hipStreamWaitEvent(stream_, wevent, 0));
#else
    return CHECK_STATUS(cudaEventSynchronize(wevent));
#endif
  }


  int deviceWait()
  {
    return CHECK_STATUS(hipStreamSynchronize(stream_));
  }

private:
  hipStream_t stream_;
  hipEvent_t event;
};

// ----------------------------------------------------------------------------
// class ConstantPolicy
// ----------------------------------------------------------------------------

#ifndef HMPP_CONSTMEM_SIZE
#define HMPP_CONSTMEM_SIZE 2048
#endif

__constant__ int64_t hmpp_constmem[HMPP_CONSTMEM_SIZE / 8];

/// Shared memory array is aligned on 64 bit thanks to that (to avoid an nvcc compilation error)
extern __shared__ int64_t hmpp_sharedmem[];

struct ConstantPolicy
{
public:
  ConstantPolicy()
  {
    static bool initialized = false;
    if(!initialized)
    {
      next_offset_ = HMPP_CONSTMEM_OFFSET;
      initialized = true;
    }
    offset_ = -1;
  }

  virtual ~ConstantPolicy()
  {

  }

  void setStaticOffset(int offset)
  {
    offset_ = offset;

    while(offset_  %  8)
        offset_ ++;
  }

  int deviceAlloc(void **ptr, size_t size)
  {
#if CUDA_VERSION >= 3020
    if( CHECK_STATUS(hipEventCreateWithFlags(&event, hipEventDisableTiming | hipEventBlockingSync)) != 0) return -1;
#else
    if( CHECK_STATUS(cudaEventCreateWithFlags(&event, cudaEventBlockingSync)) != 0) return -1;
#endif
    if(offset_ != -1)
    {
      if((offset_ + size) >= HMPP_CONSTMEM_SIZE)
        return -1;

      (*ptr) = (void *)offset_;
      return 0;
    }

    if((next_offset_ + size) >= HMPP_CONSTMEM_SIZE)
      return -1;

    (*ptr) = (void *)next_offset_;
    next_offset_ += size;
    return 0;
  }

  int deviceFree(void *ptr)
  {
    return 0;
  }

  int deviceMemcpy(void *dst,const void *src,size_t size,CopyKind kind,bool async)
  {
    size_t offset;

    switch(kind)
    {
    case HostToDevice:
      offset = (size_t)dst;
      return CHECK_STATUS(hipMemcpyToSymbol(HIP_SYMBOL(hmpp_constmem),src,size,offset,hipMemcpyHostToDevice));
    case DeviceToHost:
      offset = (size_t)src;
      return CHECK_STATUS(hipMemcpyFromSymbol(dst,HIP_SYMBOL(hmpp_constmem),size,offset,hipMemcpyDeviceToHost));
    default:
      return -1;
    }
  }

  int makeStreamWait(hipStream_t wstream)
  {
    int status;
    /* stream 0 at the moment */
    status = CHECK_STATUS(hipEventRecord(event, 0));
    if (status != 0)
      return status;
#if CUDA_VERSION >= 3020
    return CHECK_STATUS(hipStreamWaitEvent(wstream, event, 0));
#else
    return CHECK_STATUS(cudaEventSynchronize(event));
#endif
  }

  int waitOnEvent(hipEvent_t wevent)
  {
    /* stream 0 at the moment */
#if CUDA_VERSION >= 3020
    return CHECK_STATUS(hipStreamWaitEvent(0, wevent, 0));
#else
    return CHECK_STATUS(cudaEventSynchronize(wevent));
#endif
  }

  int deviceWait()
  {
    return 0;
  }

private:
  static size_t next_offset_;
  int offset_;
  hipEvent_t event;
};

size_t ConstantPolicy::next_offset_;


// ----------------------------------------------------------------------------
// class Lazy
// ----------------------------------------------------------------------------

template <typename Policy>
struct Lazy
{
  char * value;
  bool valid;
  bool allocated;
  void ** devaddr;
  Policy * policy;
  size_t size;


  Lazy(size_t elem_size)
  {
    value = new char[elem_size];
  }


  ~Lazy()
  {
    delete[] value;
  }


  int requireDeviceAlloc()
  {
    if(!allocated)
    {
      allocated = true;
      return policy->deviceAlloc(devaddr,size);
    }

    else
    {
      return 0;
    }
  }

};


// ----------------------------------------------------------------------------
// class Element
// ----------------------------------------------------------------------------

template <typename T,typename Policy>
struct Element
{
  Element(void * const * device_addr, size_t offset, Policy *policy, Lazy<Policy> * lazy)
    : device_addr_(device_addr) , offset_(offset), policy_(policy), lazy_(lazy)
  {

  }

  Element &operator=(const T & value)
  {
    if(lazy_)
    {
      *((T *)(lazy_->value)) = value;
      lazy_->valid = true;
      return *this;
    }

    if(lazy_)
      lazy_->requireDeviceAlloc();

    policy_->deviceMemcpy(((char*)(*device_addr_)) + offset_,(const char*)&value,ElemSize,HostToDevice,false);
    return *this;
  }

  Element &operator=(const Element & src)
  {
    if(src.lazy_ &&  src.lazy_->valid)
    {
      lazy_->valid = true;
      *((T *)(lazy_->value)) =  *((T *)(src.lazy_->value));
      return *this;
    }

    if(lazy_)
      lazy_->requireDeviceAlloc();
    if(src.lazy_)
      src.lazy_->requireDeviceAlloc();

    policy_->deviceMemcpy(((char*)(*device_addr_)) + offset_,((const char*)(*src.device_addr_)) + src.offset_,
                        ElemSize,DeviceToDevice,false);

    if(lazy_)
    {
      lazy_->valid = false;
    }
    return *this;
  }

  operator T()
  {
    if(lazy_ && lazy_->valid)
      return *((T *)(lazy_->value));

    T res;
    if(lazy_)
      lazy_->requireDeviceAlloc();

    policy_->deviceMemcpy(&res,((const char*)(*device_addr_)) + offset_,ElemSize,DeviceToHost,false);

    if(lazy_)
    {
      *((T *)(lazy_->value)) = res;
      lazy_->valid = true;
    }
    return res;
  }

  typedef T Type;
  enum { ElemSize = sizeof(T) };

private:
  size_t offset_;
  void *const* device_addr_;
  Policy *policy_;

public:
  Lazy<Policy> * lazy_;
};


enum DataFlags
{
    DEFAULT = 0x0,
    LAZY    = 0x1
};

// ----------------------------------------------------------------------------
// class Data
// ----------------------------------------------------------------------------

template <typename T,typename Policy>
class Data
{
public:
  typedef T Type;
  typedef Element<T,Policy> ElementType;

  enum { ElemSize = sizeof(T) };

  Data(const char * name, unsigned int flags = DEFAULT)
    : name_(name), flags_(flags),
      dim_(0), sizes_(0), size_(0),
      host_addr_(0), device_addr_(0)
  {
    policy_ = new Policy;

    if(flags_ & LAZY)
    {
      lazy_ = new Lazy<Policy>(ElemSize);
      lazy_->valid = false;
      lazy_->devaddr = 0;
      lazy_->policy = policy_;
    }
    else
      lazy_ = 0;

  }

  ~Data()
  {
    free();
    delete policy_;
    if(lazy_)
      delete lazy_;
  }

  int allocate(unsigned int dim,
               size_t idx0 = 0, size_t idx1 = 0, size_t idx2 = 0, size_t idx3 = 0,
               size_t idx4 = 0, size_t idx5 = 0, size_t idx6 = 0, size_t idx7 = 0,
               size_t idx8 = 0, size_t idx9 = 0, size_t idxA = 0, size_t idxB = 0)
  {
    const size_t sizes[] = { idx0, idx1, idx2, idx3, idx4, idx5, idx6, idx7, idx8, idx9, idxA, idxB };
    return allocate2(dim,sizes);
  }

  int allocate3(unsigned int dim_p, const size_t * sizes_p)
  {
    size_t sizes[2];

    sizes[0] = 1;
    sizes[1] = 0;
    for(int d = 0 ; d < dim_p ; d++)
    {
      sizes[0] *= sizes_p[d];
    }

    return allocate2(1, sizes);
  }

  int allocate2(unsigned int dim, const size_t * sizes)
  {
    dim_ = dim;
    sizes_ = new size_t[dim];
    dimSizes_ = new size_t[dim];

    size_ = ElemSize;
    for(int d=0;d<dim;d++)
    {
      sizes_[d] = sizes[d];
      size_ *= sizes_[d];

      size_t size = 1;
      for(int d2=d+1;d2<dim;d2++)
        size*=sizes[d2];
      dimSizes_[d] = size;
    }

    if(lazy_)
    {
      lazy_->allocated = false;
      lazy_->devaddr = &device_addr_;
      lazy_->size = size_;
      return 0;
    }
    else
      return policy_->deviceAlloc(&device_addr_,size_);
  }

  int free()
  {
    if(sizes_)
    {
      delete [] sizes_;
      delete [] dimSizes_;
      sizes_ = 0;
      dim_ = 0;
      size_ = 0;
    }

    if(device_addr_)
    {
      if(policy_->deviceFree(device_addr_) != 0)
       return -1;
      device_addr_ = 0;
    }
    return 0;
  }

  int download(void * host_addr,bool async)
  {
    if(lazy_ && lazy_->valid)
    {
      *((T *)host_addr) = *((T *)(lazy_->value));
      return 0;
    }

    if(lazy_)
    {
      lazy_->requireDeviceAlloc();
    }

    int sts = policy_->deviceMemcpy(host_addr,device_addr_,size_,DeviceToHost,async);

    if(lazy_)
    {
      lazy_->valid = true;
      *((T *)(lazy_->value)) = *((T *)host_addr);
    }

    return sts;
  }

  int upload(const void * host_addr,bool async)
  {
    if(lazy_)
    {
      lazy_->valid = true;
      *((T *)(lazy_->value)) = * ((T *)host_addr);
      lazy_->requireDeviceAlloc();
    }

    return policy_->deviceMemcpy(device_addr_,host_addr,size_,HostToDevice,async);
  }

  int downloadSection(void *host_addr,const __hmppcg_DataSection *sections,bool async)
  {
    return sectionCopy(host_addr,device_addr_,sections,DeviceToHost,async);
  }

  int uploadSection(const void *host_addr,const __hmppcg_DataSection *sections,bool async)
  {
    return sectionCopy(device_addr_,host_addr,sections,HostToDevice,async);
  }

  int makeStreamWait(hipStream_t wstream)
  {
    if(lazy_)
      lazy_->requireDeviceAlloc();
    return policy_->makeStreamWait(wstream);
  }

  int waitOnEvent(hipEvent_t wevent)
  {
    return policy_->waitOnEvent(wevent);
  }

  int waitTransfer()
  {
    return policy_->deviceWait();
  }

  ElementType operator()(size_t idx0 = 0, size_t idx1 = 0, size_t idx2 = 0, size_t idx3 = 0,
                         size_t idx4 = 0, size_t idx5 = 0, size_t idx6 = 0, size_t idx7 = 0,
                         size_t idx8 = 0, size_t idx9 = 0, size_t idxA = 0, size_t idxB = 0)
  {
    size_t sizes[] = { idx0, idx1, idx2, idx3, idx4, idx5, idx6, idx7, idx8, idx9, idxA, idxB };
    return at(sizes);
  }

  ElementType at(size_t *idx)
  {
    size_t offset = idx[0];
    return ElementType(&device_addr_,offset*ElemSize,policy_,lazy_);
  }

  template <typename Y>
  Element<Y,Policy> at(size_t offset)
  {
    return Element<Y,Policy>(&device_addr_,offset,policy_,lazy_);
  }

  ElementType operator=(const T & value)
  {
    ElementType res(&device_addr_,0,policy_,lazy_);
    res = value;
    return res;
  }

  ElementType operator=(const Data &data)
  {
    return operator=(data.value());
  }

  T value() const
  {
    ElementType res(&device_addr_,0,policy_,lazy_);
    return (T)res;
  }

  operator T()
  {
    return value();
  }

  T *getDeviceAddr()
  {
    if(lazy_)
      lazy_->requireDeviceAlloc();

    if(lazy_ && lazy_->valid)
    {
      policy_->deviceMemcpy(device_addr_,lazy_->value,size_,HostToDevice,false);
    }

    return (T*)device_addr_;
  }

  void invalidateLazy()
  {
    if(lazy_)
    {
      lazy_->valid = false;
    }
  }

private:

  Data(const Data &data) {}

  int sectionCopy(char *dst,const char *src,int offset,int cur, const __hmppcg_DataSection *sections,int lastdense,CopyKind kind,bool async)
  {
    int d;
    int size = 1;
    for(d=cur+1;d<dim_;d++)
      size *= sizes_[d];

    if(cur<(lastdense-1))
    {
      int x;
      for(x=sections[cur].from;x<=sections[cur].to;x+=sections[cur].step)
        if(sectionCopy(dst,src,offset+x*size,cur+1,sections,lastdense,kind,async) != 0)
          return -1;
    }
    else
    {
      int step = sections[cur].step;
      if(step == 1)
      {
        int start = (offset + sections[cur].from * size) * ElemSize;
        int total = (sections[cur].to - sections[cur].from + 1) * size * ElemSize;
        return policy_->deviceMemcpy(dst+start,src+start,total,kind,async);
      }
      else
      {
        int x;
        for(x=sections[cur].from;x<=sections[cur].to;x+=step)
        {
          int off = (offset + x * size) * ElemSize;
          if(policy_->deviceMemcpy(dst+off,src+off,size * ElemSize,kind,async) != 0)
            return -1;
        }
      }
    }
    return 0;
  }

  int sectionCopy(void *dst,const void *src, const __hmppcg_DataSection *sections,CopyKind kind,bool async)
  {
    int i;
    int lastdense = dim_;
    for (i = dim_ - 1 ; i >= 0 ; i --)
    {
      if ((sections[i].from == 0) && (sections[i].to == sizes_[i] - 1) && (sections[i].step == 1))
        lastdense = i;
      else
        break;
    }
    return sectionCopy((char*)dst,(const char*)src,0,0,sections,lastdense,kind,async);
  }

  const char * name_;
  size_t flags_;
  void *device_addr_;
  void *host_addr_;
  size_t dim_;
  size_t *sizes_;
  size_t *dimSizes_;
  size_t size_;

  Lazy<Policy> * lazy_;

public:
  Policy *policy_;
};

// ---------------------------------------------------------------------------
// User data
// ---------------------------------------------------------------------------
class UserData{
public:
  virtual ~UserData(){}
  UserData(){}
};

#define __HMPPCG_COMPLEX_FLOAT_DEFINED
typedef float2 __hmppcg_complex_float;

#define __HMPPCG_COMPLEX_DOUBLE_DEFINED
typedef double2 __hmppcg_complex_double;


// ---------------------------------------------------------------------------
// Allocatable Arrays
// ---------------------------------------------------------------------------
template <const size_t nb_dims> struct AArrayDesc {
  int lbounds_[nb_dims];
  size_t sizes_[nb_dims];
  size_t wholesize_;
};

#ifndef __HMPPCG_ALLOCATABLE_ARRAY_ALLOCATE
#define __HMPPCG_ALLOCATABLE_ARRAY_ALLOCATE( var, type, nb_dims, ... )                 \
        { int alloc_ranges[] = { __VA_ARGS__ };                                        \
          int hmppcg_alloc_i;                                                          \
          var ## _aarray_desc.wholesize_ = 1;                                          \
          for(hmppcg_alloc_i=0; hmppcg_alloc_i<nb_dims; hmppcg_alloc_i++){             \
            int hmppcg_alloc_first = alloc_ranges[2*hmppcg_alloc_i];                   \
            int hmppcg_alloc_last  = alloc_ranges[2*hmppcg_alloc_i + 1];               \
            int hmppcg_alloc_size  = hmppcg_alloc_last - hmppcg_alloc_first + 1;       \
            var ## _aarray_desc.lbounds_[hmppcg_alloc_i] = hmppcg_alloc_first;         \
            var ## _aarray_desc.sizes_[hmppcg_alloc_i] = hmppcg_alloc_size;            \
            var ## _aarray_desc.wholesize_ *= hmppcg_alloc_size;                       \
          }                                                                            \
          if((hmppcg_status_ = var.allocate2(nb_dims, var ## _aarray_desc.sizes_)))    \
            return;                                                                    \
        }
#endif

#ifndef __HMPPCG_ALLOCATABLE_ARRAY_DEALLOCATE
#define __HMPPCG_ALLOCATABLE_ARRAY_DEALLOCATE( var ) \
        {                                            \
          var.free();                                \
        }
#endif

#ifndef __HMPPCG_ALLOCATABLE_ARRAY_ALLOCATED
#define __HMPPCG_ALLOCATABLE_ARRAY_ALLOCATED( var ) \
        (var.getDeviceAddr() != NULL)
#endif //__HMPPCG_ALLOCATABLE_ARRAY_ALLOCATED

#ifndef __HMPPCG_ALLOCATABLE_ARRAY_WHOLESIZE
#define __HMPPCG_ALLOCATABLE_ARRAY_WHOLESIZE( var ) \
        var ## _aarray_desc.wholesize_
#endif //__HMPPCG_ALLOCATABLE_ARRAY_WHOLESIZE

#ifndef __HMPPCG_ALLOCATABLE_ARRAY_SIZE
#define __HMPPCG_ALLOCATABLE_ARRAY_SIZE( var, d ) \
        var ## _aarray_desc.sizes_[d]
#endif //__HMPPCG_ALLOCATABLE_ARRAY_SIZE

#ifndef __HMPPCG_ALLOCATABLE_ARRAY_LBOUND
#define __HMPPCG_ALLOCATABLE_ARRAY_LBOUND( var, d ) \
        var ## _aarray_desc.lbounds_[d]
#endif //__HMPPCG_ALLOCATABLE_ARRAY_LBOUND

#ifndef __HMPPCG_ALLOCATABLE_ARRAY_UBOUND
#define __HMPPCG_ALLOCATABLE_ARRAY_UBOUND( var, d ) \
        (var ## _aarray_desc.sizes_[d] + var ## _aarray_desc.lbounds_[d] - 1)
#endif //__HMPPCG_ALLOCATABLE_ARRAY_UBOUND

#ifndef __HMPP_INT_POW_FUNC
#define __HMPP_INT_POW_FUNC(func_ext_name, func_type)                             \
  __device__ func_type hmpp_pow ##func_ext_name ( func_type base, func_type exp ) \
  {                                                                               \
    if(exp < 0)                                                                   \
      return 0;                                                                   \
    func_type result = 1;                                                         \
    while (exp)                                                                   \
    {                                                                             \
      if (exp & 1)                                                                \
        result *= base;                                                           \
      exp >>= 1;                                                                  \
      base *= base;                                                               \
    }                                                                             \
    return result;                                                                \
  }
#endif

__HMPP_INT_POW_FUNC( i64, int64_t );
__HMPP_INT_POW_FUNC( i32, int32_t );
__HMPP_INT_POW_FUNC( i16, int16_t );
__HMPP_INT_POW_FUNC( i8,  int8_t );

#ifndef __HMPP_UINT_POW_FUNC
#define __HMPP_UINT_POW_FUNC(func_ext_name, func_type)                            \
  __device__ func_type hmpp_pow ##func_ext_name ( func_type base, func_type exp ) \
  {                                                                               \
    func_type result = 1;                                                         \
    while (exp)                                                                   \
    {                                                                             \
      if (exp & 1)                                                                \
        result *= base;                                                           \
      exp >>= 1;                                                                  \
      base *= base;                                                               \
    }                                                                             \
    return result;                                                                \
  }
#endif

__HMPP_UINT_POW_FUNC( ui64, uint64_t );
__HMPP_UINT_POW_FUNC( ui32, uint32_t );
__HMPP_UINT_POW_FUNC( ui16, uint16_t );
__HMPP_UINT_POW_FUNC( ui8,  uint8_t );

#endif // __HMPP_CUDADATA_H__

#ifndef __HMPPCG_COMPLEX_DOUBLE_DEFINED
#define __HMPPCG_COMPLEX_DOUBLE_DEFINED
typedef struct 
{
  double x;
  double y;
}__hmppcg_complex_double;
#endif /* __HMPPCG_COMPLEX_DOUBLE_DEFINED */

#ifndef __HMPPCG_COMPLEX_FLOAT_DEFINED
#define __HMPPCG_COMPLEX_FLOAT_DEFINED
typedef struct 
{
  float x;
  float y;
}__hmppcg_complex_float;
#endif /* __HMPPCG_COMPLEX_FLOAT_DEFINED */

template <const unsigned int blockDimX__, const unsigned int blockDimY__>
__global__ void hmpp_codelet__conv3D_loop0_(  float * HMPPCG_RESTRICT A, float * HMPPCG_RESTRICT B)
{
  int32_t k_1;
  int32_t i_1;
  k_1 = (blockDimX__ * blockIdx.x  +  threadIdx.x);
  i_1 = (blockDimY__ * blockIdx.y  +  threadIdx.y);
  bool __hmppcg_guard = (!((k_1 <= 253) & (i_1 <= 253)));
  if(__hmppcg_guard) { goto __hmppcg_label1; };
  {
    int32_t __hmppcg_end, j_1;
    for (j_1 = 0, __hmppcg_end = 64; j_1 <= __hmppcg_end; j_1 += 1)
    {
      if (j_1 <= 62)
      {
        float tmp_7;
        float tmp_8;
        float tmp_9;
        float tmp_10;
        float tmp_11;
        float tmp_12;
        float tmp_13;
        float tmp_14;
        float tmp_15;
        tmp_11 = (float) (10);
        tmp_12 = (float) (2);
        tmp_13 = (float) (5);
        tmp_14 = (float) (-8);
        tmp_15 = (float) (4);
        tmp_9 = (float) (7);
        tmp_7 = (float) (6);
        tmp_8 = (float) (-9);
        tmp_10 = (float) (-3);
        B[((((i_1 + 1) * 256) + ((4 * j_1) + 1)) * 256) + (k_1 + 1)] = ((((((((((((((tmp_12 * (A[(((i_1 * 256) + (4 * j_1)) * 256) + k_1])) + (tmp_15 * (A[((((i_1 + 2) * 256) + (4 * j_1)) * 256) + k_1]))) + (tmp_13 * (A[(((i_1 * 256) + (4 * j_1)) * 256) + k_1]))) + (tmp_9 * (A[((((i_1 + 2) * 256) + (4 * j_1)) * 256) + k_1]))) + (tmp_14 * (A[(((i_1 * 256) + (4 * j_1)) * 256) + k_1]))) + (tmp_11 * (A[((((i_1 + 2) * 256) + (4 * j_1)) * 256) + k_1]))) + (tmp_10 * (A[((((i_1 + 1) * 256) + (4 * j_1)) * 256) + (k_1 + 1)]))) + (tmp_7 * (A[((((i_1 + 1) * 256) + ((4 * j_1) + 1)) * 256) + (k_1 + 1)]))) + (tmp_8 * (A[((((i_1 + 1) * 256) + ((4 * j_1) + 2)) * 256) + (k_1 + 1)]))) + (tmp_12 * (A[(((i_1 * 256) + (4 * j_1)) * 256) + (k_1 + 2)]))) + (tmp_15 * (A[((((i_1 + 2) * 256) + (4 * j_1)) * 256) + (k_1 + 2)]))) + (tmp_13 * (A[(((i_1 * 256) + ((4 * j_1) + 1)) * 256) + (k_1 + 2)]))) + (tmp_9 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 1)) * 256) + (k_1 + 2)]))) + (tmp_14 * (A[(((i_1 * 256) + ((4 * j_1) + 2)) * 256) + (k_1 + 2)]))) + (tmp_11 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 2)) * 256) + (k_1 + 2)]));
        B[((((i_1 + 1) * 256) + ((4 * j_1) + 2)) * 256) + (k_1 + 1)] = ((((((((((((((tmp_12 * (A[(((i_1 * 256) + ((4 * j_1) + 1)) * 256) + k_1])) + (tmp_15 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 1)) * 256) + k_1]))) + (tmp_13 * (A[(((i_1 * 256) + ((4 * j_1) + 1)) * 256) + k_1]))) + (tmp_9 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 1)) * 256) + k_1]))) + (tmp_14 * (A[(((i_1 * 256) + ((4 * j_1) + 1)) * 256) + k_1]))) + (tmp_11 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 1)) * 256) + k_1]))) + (tmp_10 * (A[((((i_1 + 1) * 256) + ((4 * j_1) + 1)) * 256) + (k_1 + 1)]))) + (tmp_7 * (A[((((i_1 + 1) * 256) + ((4 * j_1) + 2)) * 256) + (k_1 + 1)]))) + (tmp_8 * (A[((((i_1 + 1) * 256) + ((4 * j_1) + 3)) * 256) + (k_1 + 1)]))) + (tmp_12 * (A[(((i_1 * 256) + ((4 * j_1) + 1)) * 256) + (k_1 + 2)]))) + (tmp_15 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 1)) * 256) + (k_1 + 2)]))) + (tmp_13 * (A[(((i_1 * 256) + ((4 * j_1) + 2)) * 256) + (k_1 + 2)]))) + (tmp_9 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 2)) * 256) + (k_1 + 2)]))) + (tmp_14 * (A[(((i_1 * 256) + ((4 * j_1) + 3)) * 256) + (k_1 + 2)]))) + (tmp_11 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 3)) * 256) + (k_1 + 2)]));
        B[((((i_1 + 1) * 256) + ((4 * j_1) + 3)) * 256) + (k_1 + 1)] = ((((((((((((((tmp_12 * (A[(((i_1 * 256) + ((4 * j_1) + 2)) * 256) + k_1])) + (tmp_15 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 2)) * 256) + k_1]))) + (tmp_13 * (A[(((i_1 * 256) + ((4 * j_1) + 2)) * 256) + k_1]))) + (tmp_9 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 2)) * 256) + k_1]))) + (tmp_14 * (A[(((i_1 * 256) + ((4 * j_1) + 2)) * 256) + k_1]))) + (tmp_11 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 2)) * 256) + k_1]))) + (tmp_10 * (A[((((i_1 + 1) * 256) + ((4 * j_1) + 2)) * 256) + (k_1 + 1)]))) + (tmp_7 * (A[((((i_1 + 1) * 256) + ((4 * j_1) + 3)) * 256) + (k_1 + 1)]))) + (tmp_8 * (A[((((i_1 + 1) * 256) + ((4 * j_1) + 4)) * 256) + (k_1 + 1)]))) + (tmp_12 * (A[(((i_1 * 256) + ((4 * j_1) + 2)) * 256) + (k_1 + 2)]))) + (tmp_15 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 2)) * 256) + (k_1 + 2)]))) + (tmp_13 * (A[(((i_1 * 256) + ((4 * j_1) + 3)) * 256) + (k_1 + 2)]))) + (tmp_9 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 3)) * 256) + (k_1 + 2)]))) + (tmp_14 * (A[(((i_1 * 256) + ((4 * j_1) + 4)) * 256) + (k_1 + 2)]))) + (tmp_11 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 4)) * 256) + (k_1 + 2)]));
        B[((((i_1 + 1) * 256) + ((4 * j_1) + 4)) * 256) + (k_1 + 1)] = ((((((((((((((tmp_12 * (A[(((i_1 * 256) + ((4 * j_1) + 3)) * 256) + k_1])) + (tmp_15 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 3)) * 256) + k_1]))) + (tmp_13 * (A[(((i_1 * 256) + ((4 * j_1) + 3)) * 256) + k_1]))) + (tmp_9 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 3)) * 256) + k_1]))) + (tmp_14 * (A[(((i_1 * 256) + ((4 * j_1) + 3)) * 256) + k_1]))) + (tmp_11 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 3)) * 256) + k_1]))) + (tmp_10 * (A[((((i_1 + 1) * 256) + ((4 * j_1) + 3)) * 256) + (k_1 + 1)]))) + (tmp_7 * (A[((((i_1 + 1) * 256) + ((4 * j_1) + 4)) * 256) + (k_1 + 1)]))) + (tmp_8 * (A[((((i_1 + 1) * 256) + ((4 * j_1) + 5)) * 256) + (k_1 + 1)]))) + (tmp_12 * (A[(((i_1 * 256) + ((4 * j_1) + 3)) * 256) + (k_1 + 2)]))) + (tmp_15 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 3)) * 256) + (k_1 + 2)]))) + (tmp_13 * (A[(((i_1 * 256) + ((4 * j_1) + 4)) * 256) + (k_1 + 2)]))) + (tmp_9 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 4)) * 256) + (k_1 + 2)]))) + (tmp_14 * (A[(((i_1 * 256) + ((4 * j_1) + 5)) * 256) + (k_1 + 2)]))) + (tmp_11 * (A[((((i_1 + 2) * 256) + ((4 * j_1) + 5)) * 256) + (k_1 + 2)]));
      } 
      else
      {
        float tmp_1__4;
        float tmp_2__4;
        float tmp_3__4;
        float tmp_4__4;
        float tmp_5__4;
        float tmp_6__4;
        tmp_1__4 = (float) (10);
        tmp_2__4 = (float) (2);
        tmp_3__4 = (float) (5);
        tmp_4__4 = (float) (-8);
        tmp_5__4 = (float) (4);
        tmp_6__4 = (float) (7);
        B[((((i_1 + 1) * 256) + (j_1 + 190)) * 256) + (k_1 + 1)] = ((((((((((((((tmp_2__4 * (A[(((i_1 * 256) + (j_1 + 189)) * 256) + k_1])) + (tmp_5__4 * (A[((((i_1 + 2) * 256) + (j_1 + 189)) * 256) + k_1]))) + (tmp_3__4 * (A[(((i_1 * 256) + (j_1 + 189)) * 256) + k_1]))) + (tmp_6__4 * (A[((((i_1 + 2) * 256) + (j_1 + 189)) * 256) + k_1]))) + (tmp_4__4 * (A[(((i_1 * 256) + (j_1 + 189)) * 256) + k_1]))) + (tmp_1__4 * (A[((((i_1 + 2) * 256) + (j_1 + 189)) * 256) + k_1]))) + (((float) (-3)) * (A[((((i_1 + 1) * 256) + (j_1 + 189)) * 256) + (k_1 + 1)]))) + (((float) (6)) * (A[((((i_1 + 1) * 256) + (j_1 + 190)) * 256) + (k_1 + 1)]))) + (((float) (-9)) * (A[((((i_1 + 1) * 256) + (j_1 + 191)) * 256) + (k_1 + 1)]))) + (tmp_2__4 * (A[(((i_1 * 256) + (j_1 + 189)) * 256) + (k_1 + 2)]))) + (tmp_5__4 * (A[((((i_1 + 2) * 256) + (j_1 + 189)) * 256) + (k_1 + 2)]))) + (tmp_3__4 * (A[(((i_1 * 256) + (j_1 + 190)) * 256) + (k_1 + 2)]))) + (tmp_6__4 * (A[((((i_1 + 2) * 256) + (j_1 + 190)) * 256) + (k_1 + 2)]))) + (tmp_4__4 * (A[(((i_1 * 256) + (j_1 + 191)) * 256) + (k_1 + 2)]))) + (tmp_1__4 * (A[((((i_1 + 2) * 256) + (j_1 + 191)) * 256) + (k_1 + 2)]));
      } 
    } 
  }
  __hmppcg_label1:;
} 

void hmpp_codelet__conv3D(  int &hmppcg_status_, void * __h, const hipDeviceProp_t &devProp, hipStream_t kernel_stream, hipEvent_t kernel_event, Data<float,DefaultPolicy> & A, Data<float,DefaultPolicy> & B)

{
  if(1LL)
  {
    unsigned int gridDimX__ = 8LL;
    HMPP_CHECK_GRID_BOUNDARY(gridDimX__);
    unsigned int gridDimY__ = 32LL;
    HMPP_CHECK_GRID_BOUNDARY(gridDimY__);
    dim3 dim_grid(gridDimX__, gridDimY__);
    const unsigned int blockDimX__ = 32LL;
    const unsigned int blockDimY__ = 8LL;
    HMPP_CHECK_BLOCK_BOUNDARY(blockDimX__*blockDimY__);
  #if CUDA_VERSION >= 3020
    A.makeStreamWait(kernel_stream);
    B.makeStreamWait(kernel_stream);
  #else
    if ((hmppcg_status_ = CHECK_STATUS(cudaThreadSynchronize()))) return;
  #endif
    dim3 dim_block(blockDimX__, blockDimY__);
    hmpp_codelet__conv3D_loop0_<blockDimX__, blockDimY__><<<dim_grid, dim_block, 0LL, kernel_stream>>>(A.getDeviceAddr(), B.getDeviceAddr());
    if ((hmppcg_status_ = CHECK_STATUS(hipGetLastError()))) return;
  #if CUDA_VERSION >= 3020
    if((hmppcg_status_ = CHECK_STATUS(hipEventRecord(kernel_event, kernel_stream)))) return;
    A.waitOnEvent(kernel_event);
    B.waitOnEvent(kernel_event);
  #else
    if ((hmppcg_status_ = CHECK_STATUS(cudaThreadSynchronize()))) return;
  #endif
    
  };
} 


// HMPP_API
#ifdef __cplusplus
#define HMPP_EXTERN extern "C"
#else
#define HMPP_EXTERN
#endif

#ifdef _WIN32
#define HMPP_EXPORT __declspec(dllexport)
#define HMPP_INLINE __inline
#else
#define HMPP_EXPORT
#define HMPP_INLINE inline
#endif

#define HMPP_API HMPP_EXTERN HMPP_EXPORT

// HMPPCG_POP_HASH
#define HMPPCG_POP_HASH(major,minor) (((major)<<16)|(minor))



// ---------------------------------------------------------------------------
// HMPP handle
// ---------------------------------------------------------------------------
typedef struct hmpp_handle_struct
{
  Data<float,DefaultPolicy> * __arg0;
  Data<float,DefaultPolicy> * __arg1;
  hipDeviceProp_t devProp;
  hipStream_t kernel_stream;
  hipEvent_t kernel_event;
  std::map<std::string,UserData*> map_user_data;
} hmpp_handle_t;


// ---------------------------------------------------------------------------
// hmpp_createInstance()
// ---------------------------------------------------------------------------
HMPP_API hmpp_handle_t * hmpp_createInstance()
{
  hmpp_handle_t * __h = new hmpp_handle_t;
  if(!__h) return 0;
  if(CHECK_STATUS(hipStreamCreate(&__h->kernel_stream)) != 0) return NULL;
  #if CUDA_VERSION >= 3020
  if(CHECK_STATUS(hipEventCreateWithFlags(&__h->kernel_event, hipEventDisableTiming | hipEventBlockingSync)) != 0) return NULL;
  #else
  if(CHECK_STATUS(cudaEventCreateWithFlags(&__h->kernel_event, cudaEventBlockingSync)) != 0) return NULL;
  #endif
  __h->__arg0 = NULL;
  __h->__arg1 = NULL;
  int device;
  hipGetDevice(&device);
  hipGetDeviceProperties(&(__h->devProp), device);
  return __h;
}

// ---------------------------------------------------------------------------
// hmpp_freeInstance()
// ---------------------------------------------------------------------------
HMPP_API int hmpp_freeInstance(hmpp_handle_t * __h)
{
  delete __h->__arg0;
  delete __h->__arg1;
  hipStreamDestroy(__h->kernel_stream);
  hipEventDestroy(__h->kernel_event);
  __h->kernel_stream = 0;
  for(std::map<std::string,UserData*>::const_iterator it = __h->map_user_data.begin(); it != __h->map_user_data.end(); it++) { delete it->second; }
  delete(__h);
  return 0;
}

// ---------------------------------------------------------------------------
// hmpp_allocateOnHWA()
// ---------------------------------------------------------------------------
HMPP_API int hmpp_allocateOnHWA(hmpp_handle_t * __h, int major, int minor, const size_t * size, size_t elsize, int dim)
{
  switch(HMPPCG_POP_HASH(major,minor))
  {
    case HMPPCG_POP_HASH(1,0): // A@hmpp_codelet__conv3D
    {
      __h->__arg0 = new Data<float,DefaultPolicy>("__arg0", DEFAULT);
      return __h->__arg0->allocate2(dim, size);
    }
    case HMPPCG_POP_HASH(1,1): // B@hmpp_codelet__conv3D
    {
      __h->__arg1 = new Data<float,DefaultPolicy>("__arg1", DEFAULT);
      return __h->__arg1->allocate2(dim, size);
    }
    default: return -1;
  }
}

HMPP_API int hmpp_allocateOutputOnHWA(hmpp_handle_t * __h, int major, int minor, const size_t * size, size_t elsize, int dim)
 { return hmpp_allocateOnHWA(__h, major, minor, size, elsize, dim); }

HMPP_API int hmpp_allocateInputOnHWA(hmpp_handle_t * __h, int major, int minor, const size_t * size, size_t elsize, int dim)
 { return hmpp_allocateOnHWA(__h, major, minor, size, elsize, dim); }

HMPP_API int hmpp_allocateInOutOnHWA(hmpp_handle_t * __h, int major, int minor, const size_t * size, size_t elsize, int dim)
 { return hmpp_allocateOnHWA(__h, major, minor, size, elsize, dim); }



// ---------------------------------------------------------------------------
// hmpp_readDataFromHWA()
// ---------------------------------------------------------------------------
HMPP_API int hmpp_readDataFromHWA(hmpp_handle_t * __h, int major, int minor, void * data, const size_t * size, size_t elsize, int dim, int async)
{
  switch(HMPPCG_POP_HASH(major,minor))
  {
    case HMPPCG_POP_HASH(1,0): // A@hmpp_codelet__conv3D
    {
      return __h->__arg0->download(data,async!=0);
    }
    case HMPPCG_POP_HASH(1,1): // B@hmpp_codelet__conv3D
    {
      return __h->__arg1->download(data,async!=0);
    }
    default: return -1;
  }
}

// ---------------------------------------------------------------------------
// hmpp_writeDataToHWA()
// ---------------------------------------------------------------------------
HMPP_API int hmpp_writeDataToHWA(hmpp_handle_t * __h, int major, int minor, const void * data, const size_t * size, size_t elsize, int dim, int async)
{
  switch(HMPPCG_POP_HASH(major,minor))
  {
    case HMPPCG_POP_HASH(1,0): // A@hmpp_codelet__conv3D
    {
      return __h->__arg0->upload(data,async!=0);
    }
    case HMPPCG_POP_HASH(1,1): // B@hmpp_codelet__conv3D
    {
      return __h->__arg1->upload(data,async!=0);
    }
    default: return -1;
  }
}

// ---------------------------------------------------------------------------
// hmpp_readDataSectionFromHWA()
// ---------------------------------------------------------------------------
HMPP_API int hmpp_readDataSectionFromHWA(hmpp_handle_t * __h, int major, int minor, void * data, const __hmppcg_DataSection *section, const size_t * size, size_t elsize, int dim, int async)
{
  switch(HMPPCG_POP_HASH(major,minor))
  {
    case HMPPCG_POP_HASH(1,0): // A@hmpp_codelet__conv3D
    {
      return __h->__arg0->downloadSection(data,section,async!=0);
    }
    case HMPPCG_POP_HASH(1,1): // B@hmpp_codelet__conv3D
    {
      return __h->__arg1->downloadSection(data,section,async!=0);
    }
    default: return -1;
  }
}

// ---------------------------------------------------------------------------
// hmpp_writeDataSectionToHWA()
// ---------------------------------------------------------------------------
HMPP_API int hmpp_writeDataSectionToHWA(hmpp_handle_t * __h, int major, int minor, const void * data, const __hmppcg_DataSection *section, const size_t * size, size_t elsize, int dim, int async)
{
  switch(HMPPCG_POP_HASH(major,minor))
  {
    case HMPPCG_POP_HASH(1,0): // A@hmpp_codelet__conv3D
    {
      return __h->__arg0->uploadSection(data,section,async!=0);
    }
    case HMPPCG_POP_HASH(1,1): // B@hmpp_codelet__conv3D
    {
      return __h->__arg1->uploadSection(data,section,async!=0);
    }
    default: return -1;
  }
}

// ---------------------------------------------------------------------------
// hmpp_waitForWriteTransfer()
// ---------------------------------------------------------------------------
HMPP_API int hmpp_waitForWriteTransfer(hmpp_handle_t * __h, int major, int minor)
{
  switch(HMPPCG_POP_HASH(major,minor))
  {
    case HMPPCG_POP_HASH(1,0): // A@hmpp_codelet__conv3D
    {
      return __h->__arg0->waitTransfer();
    }
    case HMPPCG_POP_HASH(1,1): // B@hmpp_codelet__conv3D
    {
      return __h->__arg1->waitTransfer();
    }
    default: return -1;
  }
}

// ---------------------------------------------------------------------------
// hmpp_waitForReadTransfer()
// ---------------------------------------------------------------------------
HMPP_API int hmpp_waitForReadTransfer(hmpp_handle_t * __h, int major, int minor)
{
  switch(HMPPCG_POP_HASH(major,minor))
  {
    case HMPPCG_POP_HASH(1,0): // A@hmpp_codelet__conv3D
    {
      return __h->__arg0->waitTransfer();
    }
    case HMPPCG_POP_HASH(1,1): // B@hmpp_codelet__conv3D
    {
      return __h->__arg1->waitTransfer();
    }
    default: return -1;
  }
}

// ---------------------------------------------------------------------------
// hmpp_codeletsAreReentrant()
// ---------------------------------------------------------------------------
HMPP_API int hmpp_codeletsAreReentrant()
{
  return 0;
}

// ---------------------------------------------------------------------------
// hmpp_start()
// ---------------------------------------------------------------------------
HMPP_API int hmpp_start(hmpp_handle_t * __h, int __id, int __async)
{
  int status = 0;
  switch(__id) { 
    case 1: // hmpp_codelet__conv3D(__arg0,__arg1)
      hmpp_codelet__conv3D(status, __h, __h->devProp, __h->kernel_stream, __h->kernel_event,   (*__h->__arg0), (*__h->__arg1));
      return status;
  }
  return -1;
}

// ---------------------------------------------------------------------------
// hmpp_wait()
// ---------------------------------------------------------------------------
HMPP_API int hmpp_wait(hmpp_handle_t * __h,int codelet_id)
{
  return CHECK_STATUS(hipStreamSynchronize(__h->kernel_stream));
}

// ---------------------------------------------------------------------------
// hmpp_version()
// ---------------------------------------------------------------------------
HMPP_API int hmpp_version()
{
#ifndef HMPP_RUNTIME_TARGET_VERSION
#define HMPP_RUNTIME_TARGET_VERSION(major,minor)((major << 16) | (minor << 8))
#endif
  return HMPP_RUNTIME_TARGET_VERSION(2,5);
}

//


